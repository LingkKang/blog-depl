
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <stdint.h>
#include <time.h>

const size_t HEADER_STR = 0x00; // Starting byte of BITMAPFILEHEADER.
const size_t HEADER_END = 0x0D;
const size_t HEADER_SIZE = HEADER_END - HEADER_STR + 1;
const size_t COMMON_INFO_SIZE = 40; // Common part of BITMAPINFOHEADER.

/*
 * Little-endian 32-bit integer reader.
 */
static inline uint32_t le32(const unsigned char *p)
{
    return (uint32_t)p[0] | ((uint32_t)p[1] << 8) | ((uint32_t)p[2] << 16) | ((uint32_t)p[3] << 24);
}

/*
 * Little-endian 16-bit integer reader.
 */
static inline uint16_t le16(const unsigned char *p)
{
    return (uint16_t)p[0] | ((uint16_t)p[1] << 8);
}

static inline double now_sec(void)
{
    struct timespec ts;
    clock_gettime(CLOCK_MONOTONIC, &ts);
    return (double)ts.tv_sec + (double)ts.tv_nsec * 1e-9;
}

/*
 * The CUDA kernel for pixel grayscaling. Each thread handles a specific pixel.
 */
__global__ void grayscaleKernel(unsigned char *__restrict__ img, int pix_num)
{
    int pix = (blockDim.x * blockIdx.x + threadIdx.x);
    if (pix >= pix_num)
        return;

    int i = pix * 3;
    unsigned int y = 77u * img[i + 2] + 150u * img[i + 1] + 29u * img[i];
    unsigned char gray = (unsigned char)(y >> 8);
    img[i] = gray;
    img[i + 1] = gray;
    img[i + 2] = gray;
}

int main(void)
{
    FILE *fpi = fopen("source.bmp", "rb");
    FILE *fpo = fopen("output_cuda.bmp", "wb");

    if (!fpi || !fpo)
        return 1;

    double t0 = now_sec();

    // Load header and common info header.
    unsigned char header[HEADER_SIZE];
    unsigned char common_info[COMMON_INFO_SIZE];

    (void)!fread(header, 1, HEADER_SIZE, fpi);
    (void)!fread(common_info, 1, COMMON_INFO_SIZE, fpi);
    uint32_t data_offset = le32(&header[10]);
    uint32_t remain_info_size = data_offset - HEADER_SIZE - COMMON_INFO_SIZE;
    uint32_t width = le32(&common_info[4]);
    uint32_t height = le32(&common_info[8]);
    uint16_t bpp = le16(&common_info[14]);         // Bits per pixel.
    uint16_t compression = le32(&common_info[16]); // Compression.

    if (bpp != 24 || compression != 0)
    {
        fprintf(stderr, "Unsupported BMP: expecting 24bbp, uncompressed.\n");
        return 1;
    }

    // Load remaining info header.
    unsigned char remain_info[remain_info_size];
    (void)!fread(remain_info, 1, remain_info_size, fpi);

    // Write header and info header to output image.
    fwrite(header, 1, HEADER_SIZE, fpo);
    fwrite(common_info, 1, COMMON_INFO_SIZE, fpo);
    fwrite(remain_info, 1, remain_info_size, fpo);

    uint32_t pix_num = width * height;
    uint32_t img_size = pix_num * 3u;
    unsigned char *img_h = (unsigned char *)malloc(img_size);

    double tA = now_sec();
    (void)!fread(img_h, 1, img_size, fpi);
    double tB = now_sec();
    double t_io_read = tB - tA;

    unsigned char *img_d;

    hipMalloc((void **)&img_d, img_size);
    double tC = now_sec();
    hipMemcpy(img_d, img_h, img_size, hipMemcpyHostToDevice);
    double tD = now_sec();
    double t_memcpy_h2d = tD - tC;

    int threads_per_block = (width > 1024) ? 1024 : width;
    int blocks_per_grid = (pix_num + threads_per_block - 1) / threads_per_block;
    double tE = now_sec();
    grayscaleKernel<<<blocks_per_grid, threads_per_block>>>(img_d, pix_num);
    hipDeviceSynchronize();
    double tF = now_sec();
    double t_compute = tF - tE;

    double tG = now_sec();
    hipMemcpy(img_h, img_d, img_size, hipMemcpyDeviceToHost);
    double tH = now_sec();
    double t_memcpy_d2h = tH - tG;

    hipFree(img_d);

    double tI = now_sec();
    fwrite(img_h, 1, img_size, fpo);
    double tJ = now_sec();
    double t_io_write = tJ - tI;

    free(img_h);
    fclose(fpi);
    fclose(fpo);

    double t1 = now_sec();
    double t_total = t1 - t0;

    fprintf(stderr,
            "Total: %.4f s | Read: %.4f s | H2D: %.4f s | Compute: %.4f s | D2H: %.4f s | Write: %.4f s\n",
            t_total, t_io_read, t_memcpy_h2d, t_compute, t_memcpy_d2h, t_io_write);

    return 0;
}
